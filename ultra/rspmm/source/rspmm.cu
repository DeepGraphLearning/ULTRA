#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <THC/THCAtomics.cuh>

#include "util.cuh"
#include "operator.cuh"
#include "rspmm.h"

namespace at {

// Memory & time efficient implementation of generalized spmm
// Much of the code is inspired by GE-SpMM
// https://github.com/hgyhungry/ge-spmm

namespace {

const int kCoarseningFactor = 2;
const int kThreadPerBlock = 256;

} // namespace anonymous

template <class scalar_t, class NaryOp, class BinaryOp>
__global__
void rspmm_forward_out_cuda(const int64_t *row_ptr, const int64_t *col_ind, const int64_t *layer_ind,
                            const scalar_t *weight, const scalar_t *relation, const scalar_t *input,
                            scalar_t *output,
                            int64_t num_row, int64_t nnz, int64_t dim) {
    // for best optimization, the following code is compiled with constant warpSize
    assert(blockDim.x == warpSize);

    extern __shared__ int64_t buffer[];
    int64_t *col_ind_buf = buffer;
    int64_t *layer_ind_buf = buffer + blockDim.y * warpSize;
    scalar_t *weight_buf = reinterpret_cast<scalar_t *>(layer_ind_buf + blockDim.y * warpSize);
    col_ind_buf += threadIdx.y * warpSize;
    layer_ind_buf += threadIdx.y * warpSize;
    weight_buf += threadIdx.y * warpSize;

    int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= num_row)
        return;
    int64_t d_start = blockIdx.y * warpSize * kCoarseningFactor + threadIdx.x;
    int64_t ptr_start = row_ptr[row];
    int64_t ptr_end = row + 1 < num_row ? row_ptr[row + 1] : nnz;
    scalar_t out[kCoarseningFactor];
#pragma unroll
    for (int64_t i = 0; i < kCoarseningFactor; i++)
        out[i] = NaryOp::zero;

    for (int64_t block_ptr = ptr_start; block_ptr < ptr_end; block_ptr += warpSize) {
        int64_t ptr = block_ptr + threadIdx.x;
        if (ptr < ptr_end) {
            col_ind_buf[threadIdx.x] = col_ind[ptr];
            layer_ind_buf[threadIdx.x] = layer_ind[ptr];
            weight_buf[threadIdx.x] = weight[ptr];
        }
        __syncwarp();

        int64_t max_offset = warpSize < ptr_end - block_ptr ? warpSize : ptr_end - block_ptr;
        for (int64_t offset_ptr = 0; offset_ptr < max_offset; offset_ptr++) {
            int64_t col = col_ind_buf[offset_ptr];
            int64_t layer = layer_ind_buf[offset_ptr];
            scalar_t w = weight_buf[offset_ptr];
#pragma unroll
            for (int64_t i = 0; i < kCoarseningFactor; i++) {
                int64_t d = d_start + i * warpSize;
                if (d >= dim)
                    break;
                scalar_t x = BinaryOp::forward(relation[layer * dim + d], input[col * dim + d]);
                scalar_t y = w * x;
                out[i] = NaryOp::forward(out[i], y);
            }
        }
        __syncwarp();
    }

#pragma unroll
    for (int64_t i = 0; i < kCoarseningFactor; i++) {
        int64_t d = d_start + i * warpSize;
        if (d >= dim)
            break;
        output[row * dim + d] = out[i];
    }
}

template <class scalar_t, class NaryOp, class BinaryOp>
__global__
void rspmm_backward_out_cuda(const int64_t *row_ptr, const int64_t *col_ind, const int64_t *layer_ind,
                             const scalar_t *weight, const scalar_t *relation, const scalar_t *input,
                             const scalar_t *output, const scalar_t *output_grad,
                             scalar_t *weight_grad, scalar_t *relation_grad, scalar_t *input_grad,
                             int64_t num_row, int64_t nnz, int64_t dim) {
    // for best optimization, the following code is compiled with constant warpSize
    assert(blockDim.x == warpSize);

    extern __shared__ int64_t buffer[];
    int64_t *col_ind_buf = buffer;
    int64_t *layer_ind_buf = col_ind_buf + blockDim.y * warpSize;
    scalar_t *weight_buf = reinterpret_cast<scalar_t *>(layer_ind_buf + blockDim.y * warpSize);
    col_ind_buf += threadIdx.y * warpSize;
    layer_ind_buf += threadIdx.y * warpSize;
    weight_buf += threadIdx.y * warpSize;

    int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= num_row)
        return;
    int64_t d_start = blockIdx.y * warpSize * kCoarseningFactor + threadIdx.x;
    int64_t ptr_start = row_ptr[row];
    int64_t ptr_end = row + 1 < num_row ? row_ptr[row + 1] : nnz;

    for (int64_t block_ptr = ptr_start; block_ptr < ptr_end; block_ptr += warpSize) {
        int64_t ptr = block_ptr + threadIdx.x;
        if (ptr < ptr_end) {
            col_ind_buf[threadIdx.x] = col_ind[ptr];
            layer_ind_buf[threadIdx.x] = layer_ind[ptr];
            weight_buf[threadIdx.x] = weight[ptr];
        }
        __syncwarp();

        int64_t max_offset = warpSize < ptr_end - block_ptr ? warpSize : ptr_end - block_ptr;
        for (int64_t offset_ptr = 0; offset_ptr < max_offset; offset_ptr++) {
            int64_t col = col_ind_buf[offset_ptr];
            int64_t layer = layer_ind_buf[offset_ptr];
            scalar_t w = weight_buf[offset_ptr];
            scalar_t w_grad = 0;
#pragma unroll
            for (int64_t i = 0; i < kCoarseningFactor; i++) {
                int64_t d = d_start + i * warpSize;
                if (d >= dim)
                    break;
                scalar_t rel = relation[layer * dim + d];
                scalar_t in = input[col * dim + d];
                scalar_t out = output[row * dim + d];
                scalar_t out_grad = output_grad[row * dim + d];
                scalar_t x = BinaryOp::forward(rel, in);
                scalar_t y = w * x;
                scalar_t dx_drel = BinaryOp::backward_lhs(rel, in);
                scalar_t dx_din = BinaryOp::backward_rhs(rel, in);
                scalar_t dout_dy = NaryOp::backward(out, y);
                scalar_t dy_dw = x;
                scalar_t dy_dx = w;
                w_grad += out_grad * dout_dy * dy_dw;
                atomicAdd(&relation_grad[layer * dim + d], out_grad * dout_dy * dy_dx * dx_drel);
                atomicAdd(&input_grad[col * dim + d], out_grad * dout_dy * dy_dx * dx_din);
            }
            w_grad = warp_reduce(w_grad);
            if (threadIdx.x == 0)
                atomicAdd(&weight_grad[block_ptr + offset_ptr], w_grad);
        }
        __syncwarp();
    }
}

// only relation & input require gradients
template <class scalar_t, class NaryOp, class BinaryOp>
__global__
void rspmm_backward_out_cuda(const int64_t *row_ptr, const int64_t *col_ind, const int64_t *layer_ind,
                             const scalar_t *weight, const scalar_t *relation, const scalar_t *input,
                             const scalar_t *output, const scalar_t *output_grad,
                             scalar_t *relation_grad, scalar_t *input_grad,
                             int64_t num_row, int64_t nnz, int64_t dim) {
    // for best optimization, the following code is compiled with constant warpSize
    assert(blockDim.x == warpSize);

    extern __shared__ int64_t buffer[];
    int64_t *col_ind_buf = buffer;
    int64_t *layer_ind_buf = col_ind_buf + blockDim.y * warpSize;
    scalar_t *weight_buf = reinterpret_cast<scalar_t *>(layer_ind_buf + blockDim.y * warpSize);
    col_ind_buf += threadIdx.y * warpSize;
    layer_ind_buf += threadIdx.y * warpSize;
    weight_buf += threadIdx.y * warpSize;

    int64_t row = blockIdx.x * blockDim.y + threadIdx.y;
    if (row >= num_row)
        return;
    int64_t d_start = blockIdx.y * warpSize * kCoarseningFactor + threadIdx.x;
    int64_t ptr_start = row_ptr[row];
    int64_t ptr_end = row + 1 < num_row ? row_ptr[row + 1] : nnz;

    for (int64_t block_ptr = ptr_start; block_ptr < ptr_end; block_ptr += warpSize) {
        int64_t ptr = block_ptr + threadIdx.x;
        if (ptr < ptr_end) {
            col_ind_buf[threadIdx.x] = col_ind[ptr];
            layer_ind_buf[threadIdx.x] = layer_ind[ptr];
            weight_buf[threadIdx.x] = weight[ptr];
        }
        __syncwarp();

        int64_t max_offset = warpSize < ptr_end - block_ptr ? warpSize : ptr_end - block_ptr;
        for (int64_t offset_ptr = 0; offset_ptr < max_offset; offset_ptr++) {
            int64_t col = col_ind_buf[offset_ptr];
            int64_t layer = layer_ind_buf[offset_ptr];
            scalar_t w = weight_buf[offset_ptr];
#pragma unroll
            for (int64_t i = 0; i < kCoarseningFactor; i++) {
                int64_t d = d_start + i * warpSize;
                if (d >= dim)
                    break;
                scalar_t rel = relation[layer * dim + d];
                scalar_t in = input[col * dim + d];
                scalar_t out = output[row * dim + d];
                scalar_t out_grad = output_grad[row * dim + d];
                scalar_t x = BinaryOp::forward(rel, in);
                scalar_t y = w * x;
                scalar_t dx_drel = BinaryOp::backward_lhs(rel, in);
                scalar_t dx_din = BinaryOp::backward_rhs(rel, in);
                scalar_t dout_dy = NaryOp::backward(out, y);
                scalar_t dy_dx = w;
                atomicAdd(&relation_grad[layer * dim + d], out_grad * dout_dy * dy_dx * dx_drel);
                atomicAdd(&input_grad[col * dim + d], out_grad * dout_dy * dy_dx * dx_din);
            }
        }
        __syncwarp();
    }
}

template <template<class> class NaryOp, template<class> class BinaryOp>
Tensor rspmm_forward_cuda(const Tensor &edge_index_, const Tensor &edge_type_, const Tensor &edge_weight_,
                          const Tensor &relation_, const Tensor &input_) {
    constexpr const char *fn_name = "rspmm_forward_cuda";
    TensorArg edge_index_arg(edge_index_, "edge_index", 1), edge_type_arg(edge_type_, "edge_type", 2),
              edge_weight_arg(edge_weight_, "edge_weight", 3), relation_arg(relation_, "relation", 4),
              input_arg(input_, "input", 5);

    rspmm_forward_check(fn_name, edge_index_arg, edge_type_arg, edge_weight_arg, relation_arg, input_arg);
    checkAllSameGPU(fn_name, {edge_index_arg, edge_type_arg, edge_weight_arg, relation_arg, input_arg});

    const Tensor edge_index = edge_index_.contiguous();
    const Tensor edge_type = edge_type_.contiguous();
    const Tensor edge_weight = edge_weight_.contiguous();
    const Tensor relation = relation_.contiguous();
    const Tensor input = input_.contiguous();

    int64_t nnz = edge_index.size(1);
    int64_t num_row = input.size(0);
    int64_t dim = input.size(1);
    Tensor output = at::empty({num_row, dim}, input.options());

    Tensor row_ind = edge_index.select(0, 0);
    Tensor row_ptr = ind2ptr(row_ind, num_row);
    Tensor col_ind = edge_index.select(0, 1);
    Tensor layer_ind = edge_type;

    hipSetDevice(input.get_device());
    auto stream = at::cuda::getCurrentCUDAStream();

    const int dim_per_block = 32; // warpSize
    const int num_dim_block = (dim + dim_per_block * kCoarseningFactor - 1) / (dim_per_block * kCoarseningFactor);
    const int row_per_block = kThreadPerBlock / dim_per_block;
    const int num_row_block = (num_row + row_per_block - 1) / row_per_block;

    AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rspmm_forward_cuda", [&] {
        const int memory_size = kThreadPerBlock * (sizeof(int64_t) * 2 + sizeof(scalar_t));
        rspmm_forward_out_cuda<scalar_t, NaryOp<scalar_t>, BinaryOp<scalar_t>>
            <<<dim3(num_row_block, num_dim_block), dim3(dim_per_block, row_per_block), memory_size, stream>>>(
            row_ptr.data_ptr<int64_t>(),
            col_ind.data_ptr<int64_t>(),
            layer_ind.data_ptr<int64_t>(),
            edge_weight.data_ptr<scalar_t>(),
            relation.data_ptr<scalar_t>(),
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            num_row, nnz, dim
        );
    });

    return output;
}

template <template<class> class NaryOp, template<class> class BinaryOp>
std::tuple<Tensor, Tensor, Tensor> rspmm_backward_cuda(
        const Tensor &edge_index_, const Tensor &edge_type_, const Tensor &edge_weight_,
        const Tensor &relation_, const Tensor &input_, const Tensor &output_, const Tensor &output_grad_) {
    constexpr const char *fn_name = "rspmm_backward_cuda";
    TensorArg edge_index_arg(edge_index_, "edge_index", 1), edge_type_arg(edge_type_, "edge_type", 2),
              edge_weight_arg(edge_weight_, "edge_weight", 3), relation_arg(relation_, "relation", 4),
              input_arg(input_, "input", 5), output_arg(output_, "output", 6),
              output_grad_arg(output_grad_, "output_grad", 7);

    rspmm_backward_check(fn_name, edge_index_arg, edge_type_arg, edge_weight_arg, relation_arg, input_arg,
                         output_arg, output_grad_arg);
    checkAllSameGPU(fn_name, {edge_index_arg, edge_type_arg, edge_weight_arg, relation_arg, input_arg, output_arg,
                              output_grad_arg});

    const Tensor edge_index = edge_index_.contiguous();
    const Tensor edge_type = edge_type_.contiguous();
    const Tensor edge_weight = edge_weight_.contiguous();
    const Tensor relation = relation_.contiguous();
    const Tensor input = input_.contiguous();
    const Tensor output = output_.contiguous();
    const Tensor output_grad = output_grad_.contiguous();

    int64_t nnz = edge_index.size(1);
    int64_t num_row = input.size(0);
    int64_t dim = input.size(1);
    Tensor weight_grad = at::zeros_like(edge_weight);
    Tensor relation_grad = at::zeros_like(relation);
    Tensor input_grad = at::zeros_like(input);

    Tensor row_ind = edge_index.select(0, 0);
    Tensor row_ptr = ind2ptr(row_ind, num_row);
    Tensor col_ind = edge_index.select(0, 1);
    Tensor layer_ind = edge_type;

    hipSetDevice(input.get_device());
    auto stream = at::cuda::getCurrentCUDAStream();

    const int dim_per_block = 32; // warpSize
    const int num_dim_block = (dim + dim_per_block * kCoarseningFactor - 1) / (dim_per_block * kCoarseningFactor);
    const int row_per_block = kThreadPerBlock / dim_per_block;
    const int num_row_block = (num_row + row_per_block - 1) / row_per_block;

    if (edge_weight.requires_grad())
        AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rspmm_backward_cuda", [&] {
            const int memory_size = kThreadPerBlock * (sizeof(int64_t) * 2 + sizeof(scalar_t));
            rspmm_backward_out_cuda<scalar_t, NaryOp<scalar_t>, BinaryOp<scalar_t>>
                <<<dim3(num_row_block, num_dim_block), dim3(dim_per_block, row_per_block), memory_size, stream>>>(
                row_ptr.data_ptr<int64_t>(),
                col_ind.data_ptr<int64_t>(),
                layer_ind.data_ptr<int64_t>(),
                edge_weight.data_ptr<scalar_t>(),
                relation.data_ptr<scalar_t>(),
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                output_grad.data_ptr<scalar_t>(),
                weight_grad.data_ptr<scalar_t>(),
                relation_grad.data_ptr<scalar_t>(),
                input_grad.data_ptr<scalar_t>(),
                num_row, nnz, dim
            );
        });
    else
        AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rspmm_backward_cuda", [&] {
            const int memory_size = kThreadPerBlock * (sizeof(int64_t) * 2 + sizeof(scalar_t));
            rspmm_backward_out_cuda<scalar_t, NaryOp<scalar_t>, BinaryOp<scalar_t>>
                <<<dim3(num_row_block, num_dim_block), dim3(dim_per_block, row_per_block), memory_size, stream>>>(
                row_ptr.data_ptr<int64_t>(),
                col_ind.data_ptr<int64_t>(),
                layer_ind.data_ptr<int64_t>(),
                edge_weight.data_ptr<scalar_t>(),
                relation.data_ptr<scalar_t>(),
                input.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                output_grad.data_ptr<scalar_t>(),
                relation_grad.data_ptr<scalar_t>(),
                input_grad.data_ptr<scalar_t>(),
                num_row, nnz, dim
            );
        });

    return std::make_tuple(weight_grad, relation_grad, input_grad);
}

#define DECLARE_FORWARD_IMPL(ADD, MUL, NARYOP, BINARYOP) \
    Tensor rspmm_##ADD##_##MUL##_forward_cuda(                                                            \
            const Tensor &edge_index, const Tensor &edge_type, const Tensor &edge_weight,                 \
            const Tensor &relation, const Tensor &input) {                                                \
        return rspmm_forward_cuda<NARYOP, BINARYOP>(edge_index, edge_type, edge_weight, relation, input); \
    }

#define DECLARE_BACKWARD_IMPL(ADD, MUL, NARYOP, BINARYOP) \
    std::tuple<Tensor, Tensor, Tensor> rspmm_##ADD##_##MUL##_backward_cuda(                                 \
            const Tensor &edge_index, const Tensor &edge_type, const Tensor &edge_weight,                   \
            const Tensor &relation, const Tensor &input, const Tensor &output, const Tensor &output_grad) { \
        return rspmm_backward_cuda<NARYOP, BINARYOP>(edge_index, edge_type, edge_weight, relation, input,   \
                                                     output, output_grad);                                  \
    }

DECLARE_FORWARD_IMPL(add, mul, NaryAdd, BinaryMul)
DECLARE_BACKWARD_IMPL(add, mul, NaryAdd, BinaryMul)

DECLARE_FORWARD_IMPL(min, mul, NaryMin, BinaryMul)
DECLARE_BACKWARD_IMPL(min, mul, NaryMin, BinaryMul)

DECLARE_FORWARD_IMPL(max, mul, NaryMax, BinaryMul)
DECLARE_BACKWARD_IMPL(max, mul, NaryMax, BinaryMul)

DECLARE_FORWARD_IMPL(add, add, NaryAdd, BinaryAdd)
DECLARE_BACKWARD_IMPL(add, add, NaryAdd, BinaryAdd)

DECLARE_FORWARD_IMPL(min, add, NaryMin, BinaryAdd)
DECLARE_BACKWARD_IMPL(min, add, NaryMin, BinaryAdd)

DECLARE_FORWARD_IMPL(max, add, NaryMax, BinaryAdd)
DECLARE_BACKWARD_IMPL(max, add, NaryMax, BinaryAdd)

} // namespace at